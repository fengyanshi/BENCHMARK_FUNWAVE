
#include <hip/hip_runtime.h>
#define PI 3.141592653589793238462643
#define blocDim 256
#define powOfTwo 4
#define timerCount 10
#define min(a,b) \
   ({ __typeof__ (a) _a = (a); \
       __typeof__ (b) _b = (b); \
     _a < _b ? _a : _b; })

__global__ void guts(const long int N, const long int M,const double deltaX,const double * x,double * b)
{
	__shared__ double thisBlock[blocDim];
	__shared__ long int k,l,blockPos,start;//,fullNeeded,needed;
	double FuncTemp = 0;
	long int threadPos;

	
	if(threadIdx.x==0){
		k = blockIdx.x+2;
		l = blockIdx.y;
		start = l*(N+1);
		blockPos = k+(N+1)*l;
		//fullNeeded = k+1 + (k+1)%2;
		//needed = min(fullNeeded,blocDim);
	}
	thisBlock[threadIdx.x] = 0;
	__syncthreads();

	threadPos = threadIdx.x;
	while(threadPos<=k){
		if(threadPos==0){
			FuncTemp += x[start+threadPos];
		}else if(threadPos==1){
			if(k==2){
				FuncTemp += deltaX*x[start+threadPos]/4.0;
			}else{
				FuncTemp += deltaX*x[start+threadPos]/2.0;
			}
		}else if(threadPos<(k-1)){
			FuncTemp += deltaX*x[start+threadPos];
		}else if(threadPos==(k-1)){
			FuncTemp += 3.0/4.0*deltaX*x[start+threadPos];
		}else if(threadPos==k){
			FuncTemp += deltaX*x[start+threadPos]/4.0;
		}else{
			FuncTemp += 0;
		}

		threadPos += blockDim.x;
	}
	thisBlock[threadIdx.x] = FuncTemp;
	__syncthreads();
	for(int i=blocDim/2;i>0;i=i/2){
		if(threadIdx.x<i){
			thisBlock[threadIdx.x] += thisBlock[threadIdx.x+i];
		}
		__syncthreads();
	}
	if(threadIdx.x==0){
		b[blockPos] = thisBlock[0];
	}
	
}
