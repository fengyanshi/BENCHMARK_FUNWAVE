
#include <hip/hip_runtime.h>
#define PI 3.141592653589793238462643
#define blocDim 512
#define powOfTwo 4
#define timerCount 10


// Device function used when evaluating the closed integral
__device__ double closedIntt(double s, double t, double A, double B, double C)
{
	double val;
	if(t==0){
		val = 0;
	}else{
		val = t/sqrt(A)*log(2*A*s+B*t+2*sqrt(A*(A*s*s+B*s*t+C*t*t)));
	}
	return val;
}

// Device function used when evaluating the closed integral
__device__ double closedInts(double s, double t, double A, double B, double C)
{
	double val;
	if(s==0){
		val = 0;
	}else{
		val = s/sqrt(C)*log(2*C*t+B*s+2*sqrt(C*(A*s*s+B*s*t+C*t*t)));
	}
	return val;
}

// Main GPU kernel
__global__ void guts(const long int N, const long int M,
	const double * zeta,const double * zetaX,const double * zetaY,
	const double * zetaHalf,const double * zetaXHalf,const double * zetaYHalf,
	const double * phi,const double * phiX,const double * phiHalf,const double * phiXHalf,const double * phiYHalf,
	const double * x,const double * y,const double * xHalf,const double * yHalf,
	const double * weiX,const double * weiY,const double * epsilon,const double * source, const bool * singType,const double Fr,
	const double n,double * Func)
{
	__shared__ double A, B, C,sUpper,sLower,tNegUpper,tNegLower,tPosUpper,tPosLower,xH,yH,thisBlock[blocDim];
	__shared__ long int k,l,blockPos;
	double xDiff, yNegDiff, yPosDiff,xDiffSq, yNegDiffSq, yPosDiffSq, zetaDiff,radiusSqNeg,
		radiusSqPos,K1,K2,S2,I1,I2,singInt,sqrtStuff,FuncTemp = 0;
	long int i,j,threadPos,applyBlockPos;

	// First thread initialises variables
	if(threadIdx.x==0){
		k = blockIdx.x;
		l = blockIdx.y;
		blockPos = k+(N-1)*l;	// Get position of the block for mesh half ponts
		xH = xHalf[k];
		yH = yHalf[l];
		A = 1 + zetaXHalf[blockPos]*zetaXHalf[blockPos];//
		B = 2*zetaXHalf[blockPos]*zetaYHalf[blockPos];	//
		C = 1 + zetaYHalf[blockPos]*zetaYHalf[blockPos];//
		sUpper = x[N-1]-xH;								//
		sLower = x[0]-xH;								// Calculate values needed for closed integral
		tNegUpper = y[M-1]-yH;							//
		tNegLower = y[0]-yH;							//
		tPosUpper = -y[M-1]-yH;							//
		tPosLower = -y[0]-yH;							//
	}

	// After initialising variables
	__syncthreads();


	// Have each thread sum over some values of the double integrals
	threadPos = threadIdx.x;
	i = threadPos%N;
	j = threadPos/N;
	// Each loop is one collocation point
	while(threadPos<(M*N)){		

		// Calculate nessesary values
		xDiff = x[i]-xH;
		xDiffSq = xDiff*xDiff;

		yNegDiff = y[j]-yH;
		yNegDiffSq = yNegDiff*yNegDiff;

		yPosDiff = y[j]+yH;
		yPosDiffSq = yPosDiff*yPosDiff;

		zetaDiff = zeta[threadPos]-zetaHalf[blockPos];
		
		radiusSqNeg = sqrt(xDiffSq+yNegDiffSq+zetaDiff*zetaDiff);
		radiusSqPos = sqrt(xDiffSq+yPosDiffSq+zetaDiff*zetaDiff);

		// First integral kernel function
		K1 = (zetaDiff-xDiff*zetaX[threadPos]-yNegDiff*zetaY[threadPos])/(radiusSqNeg*radiusSqNeg*radiusSqNeg)+
			(zetaDiff-xDiff*zetaX[threadPos]-yPosDiff*zetaY[threadPos])/(radiusSqPos*radiusSqPos*radiusSqPos);

		// Second integral kernel function
		K2 = 1/radiusSqNeg+1/radiusSqPos;


		S2 = 1/sqrt(A*xDiffSq+B*xDiff*yNegDiff+C*yNegDiffSq)+1/sqrt(A*xDiffSq-B*xDiff*yPosDiff+C*yPosDiffSq);

		// Calculate contributions to the first and second integrals
		I1 = weiX[i]*weiY[j]*(phi[threadPos]-phiHalf[blockPos]-xDiff)*K1;

		I2 = weiX[i]*weiY[j]*(zetaX[threadPos]*K2-zetaXHalf[blockPos]*S2);

		// Accumulate integral contributions
		FuncTemp -= I1;
		FuncTemp -= I2;
		
		// Update collocation point
		threadPos += blockDim.x;
		i = threadPos%N;
		j = threadPos/N;
	}

	// Calculation of the 16 parts to the closed integral split between differnt threads
	if(threadIdx.x==blockDim.x-1){
		singInt = closedIntt(sUpper,tNegUpper,A,B,C);
		FuncTemp -= zetaXHalf[blockPos]*singInt;

	}else if(threadIdx.x==blockDim.x-2){
		singInt = closedInts(sUpper,tNegUpper,A,B,C);
		FuncTemp -= zetaXHalf[blockPos]*singInt;

	}else if(threadIdx.x==blockDim.x-3){
		singInt = closedIntt(sLower,tNegUpper,A,B,C);
		FuncTemp += zetaXHalf[blockPos]*singInt;

	}else if(threadIdx.x==blockDim.x-4){
		singInt = closedInts(sLower,tNegUpper,A,B,C);
		FuncTemp += zetaXHalf[blockPos]*singInt;

	}else if(threadIdx.x==blockDim.x-5){
		singInt = closedIntt(sUpper,tNegLower,A,B,C);
		FuncTemp += zetaXHalf[blockPos]*singInt;

	}else if(threadIdx.x==blockDim.x-6){
		singInt = closedInts(sUpper,tNegLower,A,B,C);
		FuncTemp += zetaXHalf[blockPos]*singInt;

	}else if(threadIdx.x==blockDim.x-7){
		singInt = closedIntt(sLower,tNegLower,A,B,C);
		FuncTemp -= zetaXHalf[blockPos]*singInt;

	}else if(threadIdx.x==blockDim.x-8){
		singInt = closedInts(sLower,tNegLower,A,B,C);
		FuncTemp -= zetaXHalf[blockPos]*singInt;

	}else if(threadIdx.x==blockDim.x-9){
		singInt = closedIntt(sUpper,tPosUpper,A,B,C);
		FuncTemp += zetaXHalf[blockPos]*singInt;

	}else if(threadIdx.x==blockDim.x-10){
		singInt = closedInts(sUpper,tPosUpper,A,B,C);
		FuncTemp += zetaXHalf[blockPos]*singInt;

	}else if(threadIdx.x==blockDim.x-11){
		singInt = closedIntt(sLower,tPosUpper,A,B,C);
		FuncTemp -= zetaXHalf[blockPos]*singInt;

	}else if(threadIdx.x==blockDim.x-12){
		singInt = closedInts(sLower,tPosUpper,A,B,C);
		FuncTemp -= zetaXHalf[blockPos]*singInt;

	}else if(threadIdx.x==blockDim.x-13){
		singInt = closedIntt(sUpper,tPosLower,A,B,C);
		FuncTemp -= zetaXHalf[blockPos]*singInt;

	}else if(threadIdx.x==blockDim.x-14){
		singInt = closedInts(sUpper,tPosLower,A,B,C);
		FuncTemp -= zetaXHalf[blockPos]*singInt;

	}else if(threadIdx.x==blockDim.x-15){
		singInt = closedIntt(sLower,tPosLower,A,B,C);
		FuncTemp += zetaXHalf[blockPos]*singInt;

	}else if(threadIdx.x==blockDim.x-16){
		singInt = closedInts(sLower,tPosLower,A,B,C);
		FuncTemp += zetaXHalf[blockPos]*singInt;

	// Add the phi part of the BIE
	}else if(threadIdx.x==blockDim.x-17){
		FuncTemp += 2*PI*(phiHalf[blockPos]-xHalf[k]);

	
	}
	// Add the source contribution
	if(threadIdx.x==blockDim.x-18){
		sqrtStuff = sqrt((xH-source[0])*(xH-source[0])
					+(yH-source[1])*(yH-source[1])
					+(zetaHalf[blockPos]-source[2])*(zetaHalf[blockPos]-source[2]));
			if(singType[0]){
				FuncTemp += epsilon[0]/sqrtStuff;
			}else{
				FuncTemp -= epsilon[0]*(xH-source[0])/(sqrtStuff*sqrtStuff*sqrtStuff);
			}
	}

	// Add total contribution from thread to a storage vector
	thisBlock[threadIdx.x] = FuncTemp;

	// All threads finished evaluating the BIE
	__syncthreads();
   
	// Sum up all thread contributions
	for(i=blocDim/2;i>0;i=i/2){
		if(threadIdx.x<i){
			thisBlock[threadIdx.x] += thisBlock[threadIdx.x+i];
		}
		__syncthreads();
	}

	// Store complete BIE in correct loction of output vector
	if(threadIdx.x==0){
		applyBlockPos = k+(N+1)*l+2;
		Func[applyBlockPos+M*(N+1)] = thisBlock[0];
	}

	// Split the Bernoulli's equation and radion conditions between 4 blocks

	// Fist block calculates Bernoulli's equation
	if(k==0&&l==0){

		// Have each thread compute Bernoulli's equation for a mesh half point
		threadPos = threadIdx.x;
		while(threadPos<(M*(N-1))){
			i = threadPos%(N-1);
			j = threadPos/(N-1);
			applyBlockPos = i+(N+1)*j+2;

			// Bernoulli's Equation
			Func[applyBlockPos]=((1+zetaXHalf[threadPos]*zetaXHalf[threadPos])*phiYHalf[threadPos]*phiYHalf[threadPos]
							+(1+zetaYHalf[threadPos]*zetaYHalf[threadPos])*phiXHalf[threadPos]*phiXHalf[threadPos]
							-2*zetaXHalf[threadPos]*zetaYHalf[threadPos]*phiXHalf[threadPos]*phiYHalf[threadPos])/
							(2*(1+zetaXHalf[threadPos]*zetaXHalf[threadPos]+zetaYHalf[threadPos]*zetaYHalf[threadPos]))
							+zetaHalf[threadPos]/(Fr*Fr)-0.5;
		
			threadPos += blockDim.x;
		}

	//  Second block calculates the phi radiation condition
	}else if(k==0&&l==1){
		threadPos = threadIdx.x;
		while(threadPos<M){
			i = threadPos%(N-1);
			j = threadPos/(N-1);
			applyBlockPos = (N+1)*i;

			// phi radiation condition
			Func[applyBlockPos]=x[0]*(phiX[threadPos*N]-1)+n*(phi[threadPos*N]-x[0]);
			threadPos += blockDim.x;
		}

	//  Third block calculates the phiX radiation condition
	}else if(k==0&&l==2){
		threadPos = threadIdx.x;
		while(threadPos<M){
			i = threadPos%(N-1);
			j = threadPos/(N-1);
			applyBlockPos = (N+1)*i+1;

			// phiX radiation condition
			Func[applyBlockPos]=x[0]/(x[1]-x[0])*(phiX[threadPos*N+1]-phiX[threadPos*N])+n*(phiX[threadPos*N]-1);
			threadPos += blockDim.x;
		}

	//  Fourth block calculates the zeta radiation condition
	}else if(k==0&&l==3){
		threadPos = threadIdx.x;
		while(threadPos<M){
			i = threadPos%(N-1);
			j = threadPos/(N-1);
			applyBlockPos = (N+1)*i;

			// zeta radiation condition
			Func[applyBlockPos+M*(N+1)]=x[0]*zetaX[threadPos*N]+n*zeta[threadPos*N];
			threadPos += blockDim.x;
		}

	//  Fifth block calculates the zetaX radiation condition
	}else if(k==0&&l==4){
		threadPos = threadIdx.x;
		while(threadPos<M){
			i = threadPos%(N-1);
			j = threadPos/(N-1);
			applyBlockPos = (N+1)*i+1;

			// zetaX radiation condition
			Func[applyBlockPos+M*(N+1)]=x[0]/(x[1]-x[0])*(zetaX[threadPos*N+1]-zetaX[threadPos*N])+n*zetaX[threadPos*N];
			threadPos += blockDim.x;
		}
	}

}
